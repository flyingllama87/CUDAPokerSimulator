#include "hip/hip_runtime.h"

#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <stdio.h>
#include <iostream>
#include <string>
#include <sstream>
#include <time.h>
#include "windows.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "hip/hip_runtime_api.h"

using namespace std;

#define GRID_SIZE 16
#define BLOCK_SIZE 256
#define GAMES_PER_KERNEL 127

#define START_STACK_SIZE 500
#define SMALL_BLINDS 5

#define NUM_SUITS 4
#define NUM_CARDS 13
#define NUM_HANDS 9
#define NUM_PLAYERS 9

__device__ enum HandType { HighCard = 0, OnePair, TwoPair, ThreeOfAKind, Straight, Flush, FullHouse, FourOfAKind, StraightFlush, MaxHand };
__device__ enum CardType { c2 = 0, c3, c4, c5, c6, c7, c8, c9, cT, cJ, cQ, cK, cA, MaxCard };
__device__ enum SuitType { Club = 0, Diamond, Heart, Spade, MaxSuit };

const char* pHands[NUM_HANDS] = { "HighCard", "OnePair", "TwoPair", "ThreeOfAKind", "Straight", "Flush", "FullHouse", "FourOfAKind", "StraightFlush" };
// __device__ const char pCards[NUM_CARDS] = { '2', '3', '4', '5', '6', '7', '8', '9', 'T', 'J', 'Q', 'K', 'A' };
// __device__ const char pSuits[NUM_SUITS] = { 5, 4, 3, 6 };

int gHands[NUM_HANDS] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
int gWinnerCount[NUM_PLAYERS] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
int gWinners = 0;
int gHandsPlayed = 0;

__device__ struct Card
{
	CardType mCard;
	SuitType mSuit;

	__device__ Card() : mCard(MaxCard), mSuit(MaxSuit)
	{
	}

	__device__ void set(SuitType suit, CardType card)
	{
		mSuit = suit;
		mCard = card;
	}

	/*__device__ void print()
	{
		printf("%c%c ", pCards[mCard], pSuits[mSuit]);
	}*/

	__device__ bool operator > (const Card& c) const
	{
		return mCard > c.mCard; // Suit doesn't matter in holdem. Don't consider it.
	}

	__device__ bool operator == (const Card& c) const
	{
		return mCard == c.mCard; // Suit doesn't matter in holdem. Don't consider it.
	}

	__device__ bool operator != (const Card& c) const
	{
		return !(*this == c);
	}
};

__device__ struct Player
{
	int mStack;
	Card mCards[2];
	bool bFolded;

	__device__ Player() : mStack(0), bFolded(false)
	{
	}

	/*__device__ void print()
	{
		mCards[0].print();
		mCards[1].print();
	}*/
};

__device__ class Deck
{
	Card mCards[NUM_SUITS * NUM_CARDS];
	int mNumCards;
	hiprandState cuRandState;
	bool cuRandStateExists;

	__device__ void reset()
	{

		for (int suit = 0; suit < NUM_SUITS; ++suit)
			for (int card = 0; card < NUM_CARDS; ++card)
				mCards[(suit*NUM_CARDS) + card].set((SuitType)suit, (CardType)card);

		mNumCards = NUM_SUITS * NUM_CARDS;
	}

public:

	__device__ Card remove(int i)
	{
		//ASSERT(mNumCards > 0);

		Card removed = mCards[i];


		for (int j = i; j < mNumCards - 1; ++j)
			mCards[j] = mCards[j + 1];

		--mNumCards;
		return removed;
	}

	/*__device__ void print()
	{
		for (int suit = 0; suit < NUM_SUITS; ++suit)
			for (int card = 0; card < NUM_CARDS; ++card)
				mCards[(suit*NUM_CARDS) + card].print();

		printf("Count: %i\n", mNumCards);
	}*/

	__device__ void shuffle(int Seed)
	{
		
		Deck tmpDeck;
		int i = 0;
		int result;
		int tId = blockIdx.x * blockDim.x + threadIdx.x;

		if (cuRandStateExists == false) {
			
			hiprand_init(Seed, tId, 0, &cuRandState);
		}

	
		while (tmpDeck.mNumCards - 1)
		{
			result = hiprand(&cuRandState);
			mCards[i] = tmpDeck.remove(result % tmpDeck.mNumCards);
			++i;
		}

		mCards[i] = tmpDeck.remove(0);
		mNumCards = NUM_SUITS * NUM_CARDS;
	}

	__device__ Deck()
	{
		reset();
	}
};

__device__ struct Hand
{
	HandType mType;
	Card mCards[5];
	int mKind;
	int mKicker;
	int mKicker2;
	int mKicker3;
	bool bFolded;

	// Return 0 for equal hands, 1 if we are stronger than h, -1 if we are weaker than h.
	__device__ int isStronger(const Hand& h) const
	{
		////ASSERT(mType < MaxHand);

		if (mType != h.mType)
			return mType > h.mType ? 1 : -1;

		//ASSERT(mKind != -1 && mType < MaxHand); // Sanity check...

		if (mType == StraightFlush)
		{
			//ASSERT(mKind == 0 && mKicker == -1 && mKicker2 == -1 && mKicker3 == -1);
			//ASSERT(h.mKind == 0 && h.mKicker == -1 && h.mKicker2 == -1 && h.mKicker3 == -1);

			if (mCards[mKind] == h.mCards[h.mKind])
				return 0; // Equal straight flush (must be on the board by definition)
			else
				return mCards[mKind] > h.mCards[h.mKind] ? 1 : -1; // Evaluate who has the highest straight flush
		}
		else if (mType == FourOfAKind)
		{
			//ASSERT(mKind > -1 && mKind < 2 && mKicker > -1 && mKicker2 == -1 && mKicker3 == -1);
			//ASSERT(h.mKind > -1 && h.mKind < 2 && h.mKicker > -1 && h.mKicker2 == -1 && h.mKicker3 == -1);

			if (mCards[mKind] == h.mCards[h.mKind]) // Equal 4 of a kind
				if (mCards[mKicker] == h.mCards[h.mKicker]) // Check the kicker...
					return 0; // Equal four of a kinds, same kicker
				else
					return mCards[mKicker] > h.mCards[h.mKicker] ? 1 : -1; // Best kicker wins...
			else
				return mCards[mKind] == h.mCards[h.mKind] ? 1 : -1; // Best for a kind wins
		}
		else if (mType == Flush)
		{
			//ASSERT(mKind == 0 && mKicker == -1 && mKicker2 == -1 && mKicker3 == -1);
			//ASSERT(h.mKind == 0 && h.mKicker == -1 && h.mKicker2 == -1 && h.mKicker3 == -1);

			// Both flushes MUST be of the same suit by definition 
			// (only one suit type can ever be held at once since it requires at least 3 board cards to pull off)
			//ASSERT(mCards[mKind].mSuit == h.mCards[h.mKind].mSuit);

			// Check who has the best flush...
			if (mCards[0] == h.mCards[0] &&
				mCards[1] == h.mCards[1] &&
				mCards[2] == h.mCards[2] &&
				mCards[3] == h.mCards[3] &&
				mCards[4] == h.mCards[4])
			{
				return 0; // Equal flushes
			}
			else if (mCards[0] > h.mCards[0] ||
				mCards[1] > h.mCards[1] ||
				mCards[2] > h.mCards[2] ||
				mCards[3] > h.mCards[3] ||
				mCards[4] > h.mCards[4])
			{
				return 1; // h is the weak flush
			}
			else
			{
				return -1; // h is the strong flush
			}
		}
		else if (mType == Straight)
		{
			//ASSERT(mKind == 0 && mKicker == -1 && mKicker2 == -1 && mKicker3 == -1);
			//ASSERT(h.mKind == 0 && h.mKicker == -1 && h.mKicker2 == -1 && h.mKicker3 == -1);

			if (mCards[mKind] == h.mCards[h.mKind])
				return 0; // Equal straight
			else
				return mCards[mKind] > h.mCards[h.mKind] ? 1 : -1; // Highest straight is the strongest
		}
		else if (mType == FullHouse) // Evaluate best full house
		{
			//ASSERT(mKind > -1 && mKind < 3 && mKicker > -1 && mKicker2 == -1 && mKicker3 == -1);
			//ASSERT(h.mKind > -1 && h.mKind < 3 && h.mKicker > -1 && h.mKicker2 == -1 && h.mKicker3 == -1);

			if (mCards[mKind] == h.mCards[h.mKind]) // Do we have equal trips?
				if (mCards[mKicker] == h.mCards[h.mKicker]) // Do we also have equal pair?
					return 0; // Equal full houses
				else
					return mCards[mKicker] > h.mCards[h.mKicker] ? 1 : -1; // Hand with the greater pair wins
			else
				return mCards[mKind] > h.mCards[h.mKind] ? 1 : -1; // Hand with the greater trips wins
		}
		else if (mType == ThreeOfAKind) // Evaluate best 3 of a kind, consider first and second kickers
		{
			//ASSERT(mKind > -1 && mKind < 3 && mKicker > -1 && mKicker2 > -1 && mKicker3 == -1);
			//ASSERT(h.mKind > -1 && h.mKind < 3 && h.mKicker > -1 && h.mKicker2 > -1 && h.mKicker3 == -1);

			if (mCards[mKind] == h.mCards[h.mKind]) // Equal trips, go off the kickers
				if (mCards[mKicker] == h.mCards[h.mKicker]) // Same first kicker
					if (mCards[mKicker2] == h.mCards[h.mKicker2]) // Same second kicker
						return 0; // Exact same three of a kind
					else
						return mCards[mKicker2] > h.mCards[h.mKicker2] ? 1 : -1; // Decide who has the best hand based on the second kicker
				else
					return mCards[mKicker] > h.mCards[h.mKicker] ? 1 : -1; // Decide who has the best hand based on the first kicker
			else
				return mCards[mKind] > h.mCards[h.mKind] ? 1 : -1; // Decide who has the best hand based on who has the highest set
		}
		else if (mType == TwoPair)
		{
			//ASSERT(mKind > -1 && mKicker > -1 && mKicker2 > -1 && mKicker3 == -1);
			//ASSERT(h.mKind > -1 && h.mKicker > -1 && h.mKicker2 > -1 && h.mKicker3 == -1);

			if (mCards[mKind] == h.mCards[h.mKind]) // Same high pair
				if (mCards[mKicker] == h.mCards[h.mKicker]) // Same low pair
					if (mCards[mKicker2] == h.mCards[h.mKicker2]) // Same kicker
						return 0; // Exact same two-pair
					else
						return mCards[mKicker2] > h.mCards[h.mKicker2] ? 1 : -1; // Best kicker wins
				else
					return mCards[mKicker] > h.mCards[h.mKicker] ? 1 : -1; // Best low pair wins
			else
				return mCards[mKind] > h.mCards[h.mKind] ? 1 : -1; // Best high pair wins
		}
		else if (mType == OnePair)
		{
			//ASSERT(mKind > -1 && mKicker > -1 && mKicker2 > -1 && mKicker3 > -1);
			//ASSERT(h.mKind > -1 && h.mKicker > -1 && h.mKicker2 > -1 && h.mKicker3 > -1);

			if (mCards[mKind] == h.mCards[h.mKind]) // Same pair
				if (mCards[mKicker] == h.mCards[h.mKicker]) // Same kicker
					if (mCards[mKicker2] == h.mCards[h.mKicker2]) // Same second kicker
						if (mCards[mKicker3] == h.mCards[h.mKicker3]) // Same third kicker
							return 0; // Identical hands
						else
							return mCards[mKicker3] > h.mCards[h.mKicker3] ? 1 : -1; // Third kicker wins
					else
						return mCards[mKicker2] > h.mCards[h.mKicker2] ? 1 : -1; // Second kicker wins
				else
					return mCards[mKicker] > h.mCards[h.mKicker] ? 1 : -1; // First kicker wins
			else
				return mCards[mKind] > h.mCards[h.mKind] ? 1 : -1; // Best pair wins
		}
		else if (mType == HighCard)
		{
			//ASSERT(mKind == 0 && mKicker == -1 && mKicker2 == -1 && mKicker3 == -1);
			//ASSERT(h.mKind == 0 && h.mKicker == -1 && h.mKicker2 == -1 && h.mKicker3 == -1);

			if (mCards[mKind] == h.mCards[h.mKind]) // equal high cards
				if (mCards[1] == h.mCards[1]) // equal first kicker
					if (mCards[2] == h.mCards[2]) // equal second kicker
						if (mCards[3] == h.mCards[3]) // equal third kicker
							if (mCards[4] == h.mCards[4]) // equal forth kicker
								return 0; // Identical hands
							else
								return mCards[4] > h.mCards[4] ? 1 : -1; // Rely on the final kicker
						else
							return mCards[3] > h.mCards[3] ? 1 : -1; // Rely on the third kicker
					else
						return mCards[2] > h.mCards[2] ? 1 : -1; // Rely on the second kicker
				else
					return mCards[1] > h.mCards[1] ? 1 : -1; // Rely on the first kicker
			else
				return mCards[mKind] > h.mCards[h.mKind] ? 1 : -1; // Rely on the high card
		}

		//ASSERT(false); // We should never get down to here without already have returned a result...
		return 0;
	}

	/*__device__ void print()
	{
		for (int i = 0; i < 5; ++i)
			mCards[i].print();

		printf(" %s\n", pHands[mType]);
	}*/

	__device__ Hand() : mType(MaxHand), mKind(-1), mKicker(-1), mKicker2(-1), mKicker3(-1), bFolded(false)
	{
	}

	__device__ Hand(Card& card1, Card& card2, Card& card3, Card& card4, Card& card5) : mType(MaxHand), mKind(-1), mKicker(-1), mKicker2(-1), mKicker3(-1), bFolded(false)
	{
		// Ensure all hands are initialized
		//ASSERT(card1.mCard < MaxCard && card1.mSuit < MaxSuit &&
		//	card2.mCard < MaxCard && card2.mSuit < MaxSuit &&
		//	card3.mCard < MaxCard && card3.mSuit < MaxSuit &&
		//	card4.mCard < MaxCard && card4.mSuit < MaxSuit &&
		//	card5.mCard < MaxCard && card5.mSuit < MaxSuit);

		// Rank the cards based on value in our mCards array (this helps figure out the hand we have)
		Card tmpCard1, tmpCard2, weakCard1, weakCard2, weakCard3, weakCard4;

		if (card1 > card2) // first two cards
		{
			tmpCard1 = card1;
			weakCard1 = card2;
		}
		else
		{
			tmpCard1 = card2;
			weakCard1 = card1;
		}

		if (card3 > card4) // next two cards
		{
			tmpCard2 = card3;
			weakCard2 = card4;
		}
		else
		{
			tmpCard2 = card4;
			weakCard2 = card3;
		}

		if (tmpCard2 > tmpCard1) // determine the strongest out of the first 4 cards
		{
			weakCard3 = tmpCard1;
			tmpCard1 = tmpCard2;
		}
		else
		{
			weakCard3 = tmpCard2;
		}

		if (tmpCard1 > card5) // check against 5th card
		{
			mCards[0] = tmpCard1;
			weakCard4 = card5;
		}
		else
		{
			mCards[0] = card5;
			weakCard4 = tmpCard1;
		}

		if (weakCard1 > weakCard2) // First 2 weak cards
		{
			tmpCard1 = weakCard1;
			weakCard1 = weakCard2;
		}
		else
		{
			tmpCard1 = weakCard2;
		}

		if (weakCard3 > weakCard4) // Final 2 weak cards
		{
			tmpCard2 = weakCard3;
			weakCard2 = weakCard4;
		}
		else
		{
			tmpCard2 = weakCard4;
			weakCard2 = weakCard3;
		}

		if (tmpCard1 > tmpCard2) // set second strongest card
		{
			mCards[1] = tmpCard1;
			weakCard3 = tmpCard2;
		}
		else
		{
			mCards[1] = tmpCard2;
			weakCard3 = tmpCard1;
		}

		if (weakCard1 > weakCard2) // check the first two weak cards out of the remaining 3
		{
			tmpCard1 = weakCard1;
			weakCard1 = weakCard2;
		}
		else
		{
			tmpCard1 = weakCard2;
		}

		if (tmpCard1 > weakCard3) // check the stronger of the first two weak cards against the 3rd weak card
		{
			mCards[2] = tmpCard1; // third strongest card found

			if (weakCard1 > weakCard3) // slot the final 2 cards into position
			{
				mCards[3] = weakCard1;
				mCards[4] = weakCard3;
			}
			else
			{
				mCards[3] = weakCard3;
				mCards[4] = weakCard1;
			}
		}
		else
		{
			mCards[2] = weakCard3; // third strongest card found

			if (weakCard1 > tmpCard1) // slot the final 2 cards into position
			{
				mCards[3] = weakCard1;
				mCards[4] = tmpCard1;
			}
			else
			{
				mCards[3] = tmpCard1;
				mCards[4] = weakCard1;
			}
		}

		// 1. Check for straight
		if (mCards[0].mCard == mCards[1].mCard + 1 &&
			mCards[1].mCard == mCards[2].mCard + 1 &&
			mCards[2].mCard == mCards[3].mCard + 1 &&
			mCards[3].mCard == mCards[4].mCard + 1)
		{
			mType = Straight;
			mKind = 0;
		}

		// 2. Check for flush
		if (mCards[0].mSuit == mCards[1].mSuit &&
			mCards[1].mSuit == mCards[2].mSuit &&
			mCards[2].mSuit == mCards[3].mSuit &&
			mCards[3].mSuit == mCards[4].mSuit)
		{
			// Check for the straight flush
			if (mType == Straight)
				mType = StraightFlush;
			else
				mType = Flush;

			mKind = 0;
		}

		if (mType == Straight || mType == Flush || mType == StraightFlush)
			return;

		// 3. Check for 4 of a kind
		if (mCards[0] == mCards[1] &&
			mCards[1] == mCards[2] &&
			mCards[2] == mCards[3])
		{
			mType = FourOfAKind;
			mKind = 0;
			mKicker = 4;
			return;
		}
		else if (mCards[1] == mCards[2] &&
			mCards[2] == mCards[3] &&
			mCards[3] == mCards[4])
		{
			mType = FourOfAKind;
			mKind = 1;
			mKicker = 0;
			return;
		}

		// 4. Check for three of a kind and full houses
		if (mCards[0] == mCards[1] &&
			mCards[1] == mCards[2])
		{
			mType = ThreeOfAKind;
			mKind = 0;
			mKicker = 3;

			if (mCards[3] == mCards[4])
			{
				mType = FullHouse;
				return;
			}

			mKicker2 = 4;
			return;
		}
		else if (mCards[1] == mCards[2] &&
			mCards[2] == mCards[3])
		{
			// No possibility of a full house here...
			mType = ThreeOfAKind;
			mKind = 1;
			mKicker = 0;
			mKicker2 = 4;
			return;
		}
		else if (mCards[2] == mCards[3] &&
			mCards[3] == mCards[4])
		{
			mType = ThreeOfAKind;
			mKind = 2;
			mKicker = 0;

			if (mCards[0] == mCards[1])
			{
				mType = FullHouse;
				return;
			}

			mKicker2 = 1;
			return;
		}

		// 5. Check for two-pairs
		if (mCards[0] == mCards[1] &&
			mCards[2] == mCards[3])
		{
			mType = TwoPair;
			mKind = 0;
			mKicker = 2;
			mKicker2 = 4;
			return;
		}
		else if (mCards[1] == mCards[2] &&
			mCards[3] == mCards[4])
		{
			mType = TwoPair;
			mKind = 1;
			mKicker = 3;
			mKicker2 = 0;
			return;
		}
		else if (mCards[0] == mCards[1] &&
			mCards[3] == mCards[4])
		{
			mType = TwoPair;
			mKind = 0;
			mKicker = 3;
			mKicker2 = 2;
			return;
		}

		// 6. Check for pairs
		if (mCards[0] == mCards[1])
		{
			mType = OnePair;
			mKind = 0;
			mKicker = 2;
			mKicker2 = 3;
			mKicker3 = 4;
			return;
		}
		else if (mCards[1] == mCards[2])
		{
			mType = OnePair;
			mKind = 1;
			mKicker = 0;
			mKicker2 = 3;
			mKicker3 = 4;
			return;
		}
		else if (mCards[2] == mCards[3])
		{
			mType = OnePair;
			mKind = 2;
			mKicker = 0;
			mKicker2 = 1;
			mKicker3 = 4;
			return;
		}
		else if (mCards[3] == mCards[4])
		{
			mType = OnePair;
			mKind = 3;
			mKicker = 0;
			mKicker2 = 1;
			mKicker3 = 2;
			return;
		}

		// 7. We have just a high card...
		//ASSERT(mType == MaxHand && mKind == -1); // Sanity check
		mType = HighCard;
		mKind = 0;
	}
};

__device__ __host__ struct TableStats
{
	unsigned int mHands[NUM_HANDS];
	unsigned int mWinnerCount[NUM_PLAYERS];
	unsigned int mWinners;
	unsigned int mHandsPlayed;

	__device__ __host__ TableStats()
	{
		reset();
	}

	__device__ __host__ void reset()
	{
		mWinners = 0;
		mHandsPlayed = 0;
		memset(mHands, 0, sizeof(mHands));
		memset(mWinnerCount, 0, sizeof(mWinnerCount));
	}
};

__device__ struct Table
{
	Deck mDeck;
	Card mBoard[5];
	Player mPlayers[NUM_PLAYERS];
	Hand mHands[NUM_PLAYERS];
	int mBoardCount;
	int mNumPlayers;
	int mSmallBlind;
	int mPot;
	int mButtonPos;
	bool bWinners[NUM_PLAYERS];

	__device__ Table(int numPlayers, int startingStackSize, int smallBlind) : mBoardCount(0), mNumPlayers(numPlayers), mSmallBlind(smallBlind), mPot(0), mButtonPos(-1)
	{

		for (int i = 0; i < mNumPlayers; ++i)
		{
			mPlayers[i].mStack = startingStackSize;
			bWinners[i] = false;
		}
	}

	__device__ void resetWinners()
	{
		for (int i = 0; i < mNumPlayers; ++i)
			bWinners[i] = false;
	}

	__device__ void deal(int Seed)
	{
		mBoardCount = 0;

		// update button position
		++mButtonPos;
		if (mButtonPos >= mNumPlayers)
			mButtonPos = 0;

		// post blinds
		if (mButtonPos == mNumPlayers - 2)
		{
			mPlayers[mNumPlayers - 1].mStack -= mSmallBlind;
			mPlayers[0].mStack -= mSmallBlind * 2;
		}
		else if (mButtonPos == mNumPlayers - 1)
		{
			mPlayers[0].mStack -= mSmallBlind;
			mPlayers[1].mStack -= mSmallBlind * 2;
		}
		else
		{
			mPlayers[mButtonPos + 1].mStack -= mSmallBlind;
			mPlayers[mButtonPos + 2].mStack -= mSmallBlind * 2;
		}

		// add blinds to pot
		mPot = mSmallBlind * 3;

		// shuffle and deal
		mDeck.shuffle(Seed);
		int count = 0;
		int i = mButtonPos;
		int totalToDeal = mNumPlayers * 2;
		while (count < totalToDeal)
		{
			if (count < mNumPlayers)
				mPlayers[i].mCards[0] = mDeck.remove(0);
			else
				mPlayers[i].mCards[1] = mDeck.remove(0);

			++i;
			if (i >= mNumPlayers)
				i = 0;
			++count;
		}
	}

	__device__ void flop()
	{
		Card burn = mDeck.remove(0);

		for (int i = 0; i < 3; ++i)
			mBoard[mBoardCount++] = mDeck.remove(0);
	}

	__device__ void turn()
	{
		Card burn = mDeck.remove(0);
		mBoard[mBoardCount++] = mDeck.remove(0);
	}

	__device__ void river()
	{
		Card burn = mDeck.remove(0);
		mBoard[mBoardCount++] = mDeck.remove(0);
	}

	__device__ void evaluate(TableStats *mTableStats)
	{
		// Get best hand per player then eval best hand per player against all other hands per player
		Hand tableHand(mBoard[0], mBoard[1], mBoard[2], mBoard[3], mBoard[4]);
		
		for (int i = 0; i < mNumPlayers; ++i)
		{
			Card& card1 = mPlayers[i].mCards[0];
			Card& card2 = mPlayers[i].mCards[1];

			Hand bestHand = tableHand;

			// Try both cards (10 combinations)
			Hand tmpHand(card1, card2, mBoard[0], mBoard[1], mBoard[2]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[0], mBoard[1], mBoard[3]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[0], mBoard[1], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[0], mBoard[2], mBoard[3]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[0], mBoard[2], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[0], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[1], mBoard[2], mBoard[3]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[1], mBoard[2], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[1], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, card2, mBoard[2], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			// Try card 1 (5 combinations)
			tmpHand = Hand(card1, mBoard[0], mBoard[1], mBoard[2], mBoard[3]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, mBoard[0], mBoard[1], mBoard[2], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, mBoard[0], mBoard[1], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, mBoard[0], mBoard[2], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card1, mBoard[1], mBoard[2], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			// Try card 2 (5 combinations)
			tmpHand = Hand(card2, mBoard[0], mBoard[1], mBoard[2], mBoard[3]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card2, mBoard[0], mBoard[1], mBoard[2], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card2, mBoard[0], mBoard[1], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card2, mBoard[0], mBoard[2], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			tmpHand = Hand(card2, mBoard[1], mBoard[2], mBoard[3], mBoard[4]);
			if (bestHand.isStronger(tmpHand) == -1)
				bestHand = tmpHand;

			// Store the best hand for this player in the hands array
			mHands[i] = bestHand;
		}

		// Compare all hands against all other hands, find the winner, or winners in the case of a split pot
		resetWinners();
		int strongestHandIdx = -1;
		for (int i = 0; i < mNumPlayers; ++i)
		{
			for (int j = 0; j < mNumPlayers; ++j)
			{
				if (i < j)
				{
					int handStrength = mHands[i].isStronger(mHands[j]);
					int currStrongest = -1;

					if (handStrength == -1)
						currStrongest = j;
					else if (handStrength == 1)
						currStrongest = i;

					if (currStrongest != -1) // either i or j is stronger
					{
						if (strongestHandIdx == -1)
						{
							// we have a first strongest hand
							strongestHandIdx = currStrongest;
							bWinners[strongestHandIdx] = true;
						}
						else
						{
							handStrength = mHands[currStrongest].isStronger(mHands[strongestHandIdx]);

							if (handStrength == 0)
							{
								// currStrongest gets to share in the pot with the strongest hand
								bWinners[currStrongest] = true;
							}
							else if (handStrength == 1)
							{
								// currStrongest beats the strongest hand, currStrongest is now the strongest hand
								resetWinners();
								strongestHandIdx = currStrongest;
								bWinners[strongestHandIdx] = true;
							}
						}
					}
					else if (handStrength == 0) // i and j are equal strength
					{
						if (strongestHandIdx == -1)
						{
							// i and j are the first strongest hands, they now share the pot
							strongestHandIdx = i;
							bWinners[i] = true;
							bWinners[j] = true;
							break;
						}

						handStrength = mHands[i].isStronger(mHands[strongestHandIdx]);

						if (handStrength == 0)
						{
							// add i and j to share the pot with the stongest hand
							bWinners[i] = true;
							bWinners[j] = true;
						}
						else if (handStrength == 1)
						{
							// remove the strongest hands from the pot, add i and j who both beat it
							resetWinners();
							strongestHandIdx = i;
							bWinners[i] = true;
							bWinners[j] = true;
						}
					}
				}
			}
		}

		{
			// Accumulate statistics
			for (int i = 0; i < mNumPlayers; ++i)
			{
				++(*mTableStats).mHands[mHands[i].mType];
				if (bWinners[i])
				{
					++(*mTableStats).mWinnerCount[i];
					++(*mTableStats).mWinners;
				}
			}
			(*mTableStats).mHandsPlayed += mNumPlayers;
		}
	}
};

// Perform reduction across all warp kernels as per nVidia Kepler reduction reference
__inline__ __device__
int warpReduceSum(int val) {

	for (int offset = warpSize / 2; offset > 0; offset /= 2)
		val += __shfl_down(val, offset);
	return val;
}

// Perform Reduction across all threads in block using warm reduction as per Kepler reduction reference.
__inline__ __device__
int blockReduceSum(int val) {
	static __shared__ int shared[32];
	int lane = threadIdx.x % warpSize;
	int wid = threadIdx.x / warpSize;
	val = warpReduceSum(val);

	//write reduced value to shared memory
	if (lane == 0) shared[wid] = val;
	__syncthreads();

	//ensure we only grab a value from shared memory if that warp existed
	val = (threadIdx.x<blockDim.x / warpSize) ? shared[lane] : int(0);
	if (wid == 0) val = warpReduceSum(val);

	return val;
}

__global__ void RunGame(TableStats *ts, int Seed)
{
	Table t(NUM_PLAYERS, START_STACK_SIZE, SMALL_BLINDS);
	TableStats TableStats;

	// int tId = threadIdx.x + (blockIdx.x * blockDim.x);

	// Run Poker Game as many times as define in GAMES_PER_KERNEL times and accumulate stats in per thread table stats array
	for (int i = 0; i <= GAMES_PER_KERNEL; i++)
	{
		t.deal(Seed);
		t.flop();
		t.turn();
		t.river();
		t.evaluate(&TableStats);
	}


	// Perform sum of all 'table stats' and save answer to the first thread of each block.
	TableStats.mWinners = blockReduceSum(TableStats.mWinners);
	TableStats.mHandsPlayed = blockReduceSum(TableStats.mHandsPlayed);

	for (int c = 0; c < NUM_PLAYERS; c++)
	{
		TableStats.mWinnerCount[c] = blockReduceSum(TableStats.mWinnerCount[c]);
		TableStats.mHands[c] = blockReduceSum(TableStats.mHands[c]);
	}
			//
	if (threadIdx.x == 0) {
		ts[blockIdx.x].mWinners = TableStats.mWinners;
		ts[blockIdx.x].mHandsPlayed = TableStats.mHandsPlayed;

		for (int c = 0; c < NUM_PLAYERS; c++)
		{
			ts[blockIdx.x].mWinnerCount[c] = TableStats.mWinnerCount[c];
			ts[blockIdx.x].mHands[c] = TableStats.mHands[c];
		}
	}
}

int main()
{
	TableStats *ts;
	int TableStatMemorySize = GRID_SIZE * sizeof(TableStats);
	int Seed = (int)time(NULL);

	hipProfilerStart();

	//Allocate memory on the GPU for game results and initialize
	hipMallocManaged(&ts, TableStatMemorySize);
	hipMemset(ts, 0, TableStatMemorySize);

	printf("About to run CUDA poker simulation of %d blocks, %d threads per block for a total of %d threads each running %d games.\n", GRID_SIZE, BLOCK_SIZE, GRID_SIZE * BLOCK_SIZE, GAMES_PER_KERNEL, GRID_SIZE * BLOCK_SIZE * GAMES_PER_KERNEL);

	//for (int i = 0; i < 10; i++)
	//{
		RunGame << < GRID_SIZE, BLOCK_SIZE >> >(ts, Seed);
		hipDeviceSynchronize();
	//}


	// check for CUDA error
	hipError_t error3 = hipGetLastError();
	if (error3 != hipSuccess)
	{
		// print the CUDA error message 
		printf("CUDA error: %s\n", hipGetErrorString(error3));
	}

	printf("\nExecuted poker simulation.  About to perform sum of results and print.\n");

	// Perform reduction of results back from GPU on the CPU
	for (int i = 0; (float)i < GRID_SIZE; i++)
	{
		gWinners += ts[i].mWinners;
		gHandsPlayed += ts[i].mHandsPlayed;

		for (unsigned int c = 0; c < NUM_PLAYERS; c++) {
			gWinnerCount[c] += ts[i].mWinnerCount[c];
			gHands[c] += ts[i].mHands[c];
		}
	}

	// Print results
	printf("\nTotal Winners %d", gWinners);
	printf("\nTotal Hands Played is %d\n", gHandsPlayed);
	for (unsigned int c = 0; c < NUM_PLAYERS; c++)
		printf("\nPlayer %d has won %d the times (%.2f percent)", c + 1, gWinnerCount[c], ((float)gWinnerCount[c] / (float)gWinners) * 100.f);

	printf("\n\n");

	float total = 0;
	for (int c = 0; c < NUM_HANDS; c++)
	{
		const float percentage = ((float)gHands[c] / (float)gHandsPlayed) * 100.f;
		printf("%15s: %8.4f hit %d times\n", pHands[c], percentage, gHands[c]);
		total += percentage;
	}

	printf("\n%.2f percent | %d hands played | %d games played\n", total, gHandsPlayed, gHandsPlayed / NUM_PLAYERS);

	
	//printf("\nSimulation performed on Device %d: \"%s\"\n", 0, deviceProp.name);
	//printf("\nDevice ClockRate: \"%d\" hz\n", deviceProp.clockRate * 1000);

	hipDeviceReset();
	hipProfilerStop();

	return 0;
}
